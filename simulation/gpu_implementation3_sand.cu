#include "hip/hip_runtime.h"
#include "gpu_implementation3_sand.h"
#include "parameters_sim.h"
#include "point.h"
#include <stdio.h>
#include <iostream>
#include <vector>

#include <Eigen/Core>
#include <Eigen/LU>

#include <spdlog/spdlog.h>

#include "helper_math.cuh"

__device__ int gpu_error_indicator;
__constant__ icy::SimParams gprms;




void GPU_Implementation3::initialize()
{
    if(initialized) return;
    hipError_t err;

    int deviceCount = 0;
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) throw std::runtime_error("GPU_Implementation3::initialize() cuda error");
    if(deviceCount == 0) throw std::runtime_error("No avaialble CUDA devices");
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    spdlog::info("Compute capability {}.{}",deviceProp.major, deviceProp.minor);

    hipEventCreate(&eventCycleStart);
    hipEventCreate(&eventCycleStop);

    err = hipStreamCreate(&streamCompute);
    if(err != hipSuccess) throw std::runtime_error("GPU_Implementation3::initialize() hipEventCreate");

    initialized = true;
}

void GPU_Implementation3::cuda_update_constants()
{
    hipError_t err;
    err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_error_indicator), &error_code, sizeof(int));
    if(err != hipSuccess)
    {
        spdlog::critical("hipMemcpyToSymbol error code {}",err);
        throw std::runtime_error("gpu_error_indicator initialization");
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(gprms), prms, sizeof(icy::SimParams));
    if(err!=hipSuccess) throw std::runtime_error("cuda_update_constants: gprms");
    std::cout << "CUDA constants copied to device\n";
}

void GPU_Implementation3::cuda_allocate_arrays(size_t nGridNodes, size_t nPoints)
{
    if(!initialized) initialize();
//    size_t nGridNodes = prms->GridX*prms->GridY;
//    size_t nPoints = prms->nPts;
    hipError_t err;

    // device memory for grid
    hipFree(prms->grid_array);
    hipFree(prms->pts_array);
    hipHostFree(tmp_transfer_buffer);

    err = hipMallocPitch (&prms->grid_array, &prms->nGridPitch, sizeof(real)*nGridNodes, icy::SimParams::nGridArrays);
    if(err != hipSuccess) throw std::runtime_error("cuda_allocate_arrays");
    spdlog::info("Grid: requested {}B, pitched width is {} B", sizeof(real)*nGridNodes, prms->nGridPitch);

    // device memory for points
    err = hipMallocPitch (&prms->pts_array, &prms->nPtsPitch, sizeof(real)*nPoints, icy::SimParams::nPtsArrays);
    if(err != hipSuccess) throw std::runtime_error("cuda_allocate_arrays");
    spdlog::info("Points: requested {} B, pitched width is {} B", sizeof(real)*nPoints, prms->nPtsPitch);
    spdlog::info("cuda_allocate_arrays done");

    // pinned host memory
    err = hipHostMalloc(&tmp_transfer_buffer, prms->nPtsPitch*icy::SimParams::nPtsArrays);
    if(err!=hipSuccess) throw std::runtime_error("GPU_Implementation3::Prepare(int nPoints)");

    double MemAllocGrid = (double)prms->nGridPitch*icy::SimParams::nGridArrays/(1024*1024);
    double MemAllocPoints = (double)prms->nPtsPitch*icy::SimParams::nPtsArrays/(1024*1024);
    double MemAllocTotal = MemAllocGrid + MemAllocPoints;
    spdlog::info("memory use: grid {:03.2f} Mb; points {:03.2f} Mb ; total {:03.2f} Mb",
                 MemAllocGrid, MemAllocPoints, MemAllocTotal);
    error_code = 0;
}

void GPU_Implementation3::transfer_ponts_to_device(const std::vector<icy::Point> &points)
{
    int n = prms->nPtsPitch/sizeof(real);

    for(int i=0;i<prms->nPts;i++)
    {
        tmp_transfer_buffer[i + n*icy::SimParams::posx] = points[i].pos[0];
        tmp_transfer_buffer[i + n*icy::SimParams::posy] = points[i].pos[1];
        tmp_transfer_buffer[i + n*icy::SimParams::velx] = points[i].velocity[0];
        tmp_transfer_buffer[i + n*icy::SimParams::vely] = points[i].velocity[1];
        tmp_transfer_buffer[i + n*icy::SimParams::Bp00] = points[i].Bp(0,0);
        tmp_transfer_buffer[i + n*icy::SimParams::Bp01] = points[i].Bp(0,1);
        tmp_transfer_buffer[i + n*icy::SimParams::Bp10] = points[i].Bp(1,0);
        tmp_transfer_buffer[i + n*icy::SimParams::Bp11] = points[i].Bp(1,1);
        tmp_transfer_buffer[i + n*icy::SimParams::Fe00] = points[i].Fe(0,0);
        tmp_transfer_buffer[i + n*icy::SimParams::Fe01] = points[i].Fe(0,1);
        tmp_transfer_buffer[i + n*icy::SimParams::Fe10] = points[i].Fe(1,0);
        tmp_transfer_buffer[i + n*icy::SimParams::Fe11] = points[i].Fe(1,1);
        tmp_transfer_buffer[i + n*icy::SimParams::idx_case] = points[i].q;
        tmp_transfer_buffer[i + n*icy::SimParams::idx_Jp] = points[i].Jp_inv;
        tmp_transfer_buffer[i + n*icy::SimParams::idx_zeta] = points[i].zeta;
    }

    // transfer point data to device
    hipError_t err;
    err = hipMemcpy(prms->pts_array, tmp_transfer_buffer, prms->nPtsPitch*icy::SimParams::nPtsArrays, hipMemcpyHostToDevice);
    if(err != hipSuccess) throw std::runtime_error("transfer_points_to_device");
}

void GPU_Implementation3::cuda_transfer_from_device()
{
    hipError_t err;

    err = hipMemcpyAsync(tmp_transfer_buffer, prms->pts_array, prms->nPtsPitch*icy::SimParams::nPtsArrays,
                          hipMemcpyDeviceToHost, streamCompute);
    if(err != hipSuccess) throw std::runtime_error("cuda_transfer_from_device");

    err = hipMemcpyFromSymbolAsync(&error_code, HIP_SYMBOL(gpu_error_indicator), sizeof(int), 0, hipMemcpyDeviceToHost, streamCompute);
    if(err != hipSuccess)
    {
        std::cout << "cuda_p2g hipMemcpyFromSymbol error\n";
        throw std::runtime_error("cuda_transfer_from_device");
    }

    void* userData = reinterpret_cast<void*>(this);
    hipStreamAddCallback(streamCompute, GPU_Implementation3::callback_transfer_from_device_completion, userData, 0);
}

void CUDART_CB GPU_Implementation3::callback_transfer_from_device_completion(hipStream_t stream, hipError_t status, void *userData)
{
    // simulation data was copied to host memory -> proceed with processing of this data
    GPU_Implementation3 *m = reinterpret_cast<GPU_Implementation3*>(userData);
    if(m->transfer_completion_callback) m->transfer_completion_callback();
}

void GPU_Implementation3::transfer_ponts_to_host_finalize(std::vector<icy::Point> &points)
{
    int n = prms->nPtsPitch/sizeof(real);
    if(points.size() != prms->nPts) points.resize(prms->nPts);
    for(int i=0;i<prms->nPts;i++)
    {
        points[i].pos[0] = tmp_transfer_buffer[i + n*icy::SimParams::posx];
        points[i].pos[1] = tmp_transfer_buffer[i + n*icy::SimParams::posy];
        points[i].velocity[0] = tmp_transfer_buffer[i + n*icy::SimParams::velx];
        points[i].velocity[1] = tmp_transfer_buffer[i + n*icy::SimParams::vely];
        points[i].Bp(0,0) = tmp_transfer_buffer[i + n*icy::SimParams::Bp00];
        points[i].Bp(0,1) = tmp_transfer_buffer[i + n*icy::SimParams::Bp01];
        points[i].Bp(1,0) = tmp_transfer_buffer[i + n*icy::SimParams::Bp10];
        points[i].Bp(1,1) = tmp_transfer_buffer[i + n*icy::SimParams::Bp11];
        points[i].Fe(0,0) = tmp_transfer_buffer[i + n*icy::SimParams::Fe00];
        points[i].Fe(0,1) = tmp_transfer_buffer[i + n*icy::SimParams::Fe01];
        points[i].Fe(1,0) = tmp_transfer_buffer[i + n*icy::SimParams::Fe10];
        points[i].Fe(1,1) = tmp_transfer_buffer[i + n*icy::SimParams::Fe11];
        points[i].Jp_inv = tmp_transfer_buffer[i + n*icy::SimParams::idx_Jp];
        points[i].zeta = tmp_transfer_buffer[i + n*icy::SimParams::idx_zeta];

        points[i].visualize_p = tmp_transfer_buffer[i + n*icy::SimParams::idx_p];
        points[i].visualize_p0 = tmp_transfer_buffer[i + n*icy::SimParams::idx_p0];
        points[i].visualize_q = tmp_transfer_buffer[i + n*icy::SimParams::idx_q];
        points[i].visualize_psi = tmp_transfer_buffer[i + n*icy::SimParams::idx_psi];
        points[i].q = tmp_transfer_buffer[i + n*icy::SimParams::idx_case];
    }
}


void GPU_Implementation3::cuda_reset_grid()
{
    hipError_t err = hipMemsetAsync(prms->grid_array, 0, prms->nGridPitch*icy::SimParams::nGridArrays, streamCompute);
    if(err != hipSuccess) throw std::runtime_error("cuda_reset_grid error");
}

void GPU_Implementation3::cuda_p2g()
{
    const int nPoints = prms->nPts;
    hipError_t err;

    int tpb = prms->tpb_P2G;
    int blocksPerGrid = (nPoints + tpb - 1) / tpb;
    v2_kernel_p2g<<<blocksPerGrid, tpb, 0, streamCompute>>>();
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        std::cout << "cuda_p2g error executing kernel_p2g " << err << std::endl;
        throw std::runtime_error("cuda_p2g");
    }
}

void GPU_Implementation3::cuda_update_nodes(real indenter_x, real indenter_y)
{
    const int nGridNodes = prms->GridX*prms->GridY;
    hipError_t err;
    int tpb = prms->tpb_Upd;
    int blocksPerGrid = (nGridNodes + tpb - 1) / tpb;
    v2_kernel_update_nodes<<<blocksPerGrid, tpb, 0, streamCompute>>>(indenter_x, indenter_y);
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        std::cout << "cuda_update_nodes\n";
        throw std::runtime_error("cuda_update_nodes");
    }
}

void GPU_Implementation3::cuda_g2p()
{
    const int nPoints = prms->nPts;
    hipError_t err;
    int tpb = prms->tpb_G2P;
    int blocksPerGrid = (nPoints + tpb - 1) / tpb;
    v2_kernel_g2p<<<blocksPerGrid, tpb, 0, streamCompute>>>();
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        std::cout << "cuda_g2p error " << err << '\n';
        throw std::runtime_error("cuda_g2p");
    }
}

// ==============================  Functions that compute Kirchhoff stress via Strain Energy Density ========




// ==============================  kernels  ====================================

__global__ void v2_kernel_p2g()
{
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int &nPoints = gprms.nPts;
    if(pt_idx >= nPoints) return;

    const real &dt = gprms.InitialTimeStep;
    const real &vol = gprms.ParticleVolume;
    const real &h = gprms.cellsize;
    const real &h_inv = gprms.cellsize_inv;
    const real &Dinv = gprms.Dp_inv;
//    real lambda = gprms.lambda;
//    real mu = gprms.mu;
    const int &gridX = gprms.GridX;
    const int &gridY = gprms.GridY;
    const real &particle_mass = gprms.ParticleMass;
    const int &nGridPitch = gprms.nGridPitch/sizeof(real);
    const int nPtsPitch = gprms.nPtsPitch/sizeof(real);

    icy::Point p;
    p.pos[0] = gprms.pts_array[icy::SimParams::posx*nPtsPitch + pt_idx];
    p.pos[1] = gprms.pts_array[icy::SimParams::posy*nPtsPitch + pt_idx];
    p.velocity[0] = gprms.pts_array[icy::SimParams::velx*nPtsPitch + pt_idx];
    p.velocity[1] = gprms.pts_array[icy::SimParams::vely*nPtsPitch + pt_idx];
    p.Bp(0,0) = gprms.pts_array[icy::SimParams::Bp00*nPtsPitch + pt_idx];
    p.Bp(0,1) = gprms.pts_array[icy::SimParams::Bp01*nPtsPitch + pt_idx];
    p.Bp(1,0) = gprms.pts_array[icy::SimParams::Bp10*nPtsPitch + pt_idx];
    p.Bp(1,1) = gprms.pts_array[icy::SimParams::Bp11*nPtsPitch + pt_idx];
    p.Fe(0,0) = gprms.pts_array[icy::SimParams::Fe00*nPtsPitch + pt_idx];
    p.Fe(0,1) = gprms.pts_array[icy::SimParams::Fe01*nPtsPitch + pt_idx];
    p.Fe(1,0) = gprms.pts_array[icy::SimParams::Fe10*nPtsPitch + pt_idx];
    p.Fe(1,1) = gprms.pts_array[icy::SimParams::Fe11*nPtsPitch + pt_idx];
    p.Jp_inv = gprms.pts_array[icy::SimParams::idx_Jp*nPtsPitch + pt_idx];
    p.zeta = gprms.pts_array[icy::SimParams::idx_zeta*nPtsPitch + pt_idx];



    Matrix2r PFt = KirchhoffStress_Wolper(p.Fe, p.zeta, p.Jp_inv);

    Matrix2r subterm2 = particle_mass*p.Bp - (dt*vol*Dinv)*PFt;

    constexpr real offset = 0.5;  // 0 for cubic; 0.5 for quadratic
    const int i0 = (int)(p.pos[0]*h_inv - offset);
    const int j0 = (int)(p.pos[1]*h_inv - offset);

    Vector2r base_coord(i0,j0);
    Vector2r fx = p.pos*h_inv - base_coord;

    real v0[2] {1.5-fx[0], 1.5-fx[1]};
    real v1[2] {fx[0]-1.,  fx[1]-1.};
    real v2[2] {fx[0]-.5,  fx[1]-.5};

    real w[3][2] = {{.5*v0[0]*v0[0],  .5*v0[1]*v0[1]},
                    {.75-v1[0]*v1[0], .75-v1[1]*v1[1]},
                    {.5*v2[0]*v2[0],  .5*v2[1]*v2[1]}};

    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
        {
            real Wip = w[i][0]*w[j][1];
            Vector2r dpos((i-fx[0])*h, (j-fx[1])*h);
            Vector2r incV = Wip*(p.velocity*particle_mass + subterm2*dpos);
            real incM = Wip*particle_mass;

            int idx_gridnode = (i+i0) + (j+j0)*gridX;
            if((i+i0) < 0 || (j+j0) < 0 || (i+i0) >=gridX || (j+j0)>=gridY) gpu_error_indicator = 1;

            // Udpate mass, velocity and force
            atomicAdd(&gprms.grid_array[0*nGridPitch + idx_gridnode], incM);
            atomicAdd(&gprms.grid_array[1*nGridPitch + idx_gridnode], incV[0]);
            atomicAdd(&gprms.grid_array[2*nGridPitch + idx_gridnode], incV[1]);
        }
}

__global__ void v2_kernel_update_nodes(real indenter_x, real indenter_y)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int &nGridNodes = gprms.GridX*gprms.GridY;
    if(idx >= nGridNodes) return;

    const int &nGridPitch = gprms.nGridPitch/sizeof(real);
    real mass = gprms.grid_array[0*nGridPitch + idx];
    if(mass == 0) return;

    Vector2r velocity;
    velocity[0] = gprms.grid_array[1*nGridPitch + idx];
    velocity[1] = gprms.grid_array[2*nGridPitch + idx];

    const real &gravity = gprms.Gravity;
    const real &indRsq = gprms.IndRSq;
    const int &gridX = gprms.GridX;
    const int &gridY = gprms.GridY;
    const real &dt = gprms.InitialTimeStep;
    const real &ind_velocity = gprms.IndVelocity;
    const real &cellsize = gprms.cellsize;
    const real &ice_friction_coeff = gprms.IceFrictionCoefficient;

    const Vector2r vco(ind_velocity,0);  // velocity of the collision object (indenter)
    const Vector2r indCenter(indenter_x, indenter_y);

    velocity /= mass;
    velocity[1] -= dt*gravity;
    real vmax = 0.5*cellsize/dt;
    if(velocity.norm() > vmax) velocity = velocity/velocity.norm()*vmax;

    int idx_x = idx % gridX;
    int idx_y = idx / gridX;

    // indenter
    Vector2r gnpos(idx_x*cellsize, idx_y*cellsize);
    Vector2r n = gnpos - indCenter;
    if(n.squaredNorm() < indRsq)
    {
        // grid node is inside the indenter
        Vector2r vrel = velocity - vco;
        n.normalize();
        real vn = vrel.dot(n);   // normal component of the velocity
        if(vn < 0)
        {
            Vector2r vt = vrel - n*vn;   // tangential portion of relative velocity
            velocity = vco + vt + ice_friction_coeff*vn*vt.normalized();
        }
    }

    // attached bottom layer
    if(idx_y <= 3) velocity.setZero();
    else if(idx_y >= gridY-4 && velocity[1]>0) velocity[1] = 0;
    if(idx_x <= 3 && velocity.x()<0) velocity[0] = 0;
    else if(idx_x >= gridX-5) velocity[0] = 0;
    if(gprms.HoldBlockOnTheRight==1)
    {
        int blocksGridX = gprms.BlockLength*gprms.cellsize_inv+5-2;
        if(idx_x >= blocksGridX) velocity.setZero();
    }
    else if(gprms.HoldBlockOnTheRight==2)
    {
        int blocksGridX = gprms.BlockLength*gprms.cellsize_inv+5-2;
        int blocksGridY = gprms.BlockHeight/2*gprms.cellsize_inv+2;
        if(idx_x >= blocksGridX && idx_x <= blocksGridX + 2 && idx_y < blocksGridY) velocity.setZero();
        if(idx_x <= 7 && idx_x > 4 && idx_y < blocksGridY) velocity.setZero();
    }


    // write the updated grid velocity back to memory
    gprms.grid_array[1*nGridPitch + idx] = velocity[0];
    gprms.grid_array[2*nGridPitch + idx] = velocity[1];
}

__global__ void v2_kernel_g2p()
{
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int &nPoints = gprms.nPts;
    if(pt_idx >= nPoints) return;

    const int nPtsPitched = gprms.nPtsPitch/sizeof(real);
    const int nGridPitched = gprms.nGridPitch/sizeof(real);
    const real &h_inv = gprms.cellsize_inv;
    const real &dt = gprms.InitialTimeStep;
    const int &gridX = gprms.GridX;

    icy::Point p;
    p.pos[0] = gprms.pts_array[icy::SimParams::posx*nPtsPitched + pt_idx];
    p.pos[1] = gprms.pts_array[icy::SimParams::posy*nPtsPitched + pt_idx];
    p.Fe(0,0) = gprms.pts_array[icy::SimParams::Fe00*nPtsPitched + pt_idx];
    p.Fe(0,1) = gprms.pts_array[icy::SimParams::Fe01*nPtsPitched + pt_idx];
    p.Fe(1,0) = gprms.pts_array[icy::SimParams::Fe10*nPtsPitched + pt_idx];
    p.Fe(1,1) = gprms.pts_array[icy::SimParams::Fe11*nPtsPitched + pt_idx];

    p.q = gprms.pts_array[icy::SimParams::idx_case*nPtsPitched + pt_idx];
    p.Jp_inv = gprms.pts_array[icy::SimParams::idx_Jp*nPtsPitched + pt_idx];
    p.zeta = gprms.pts_array[icy::SimParams::idx_zeta*nPtsPitched + pt_idx];

    p.velocity.setZero();
    p.Bp.setZero();

    constexpr real offset = 0.5;  // 0 for cubic; 0.5 for quadratic
    const int i0 = (int)(p.pos[0]*h_inv - offset);
    const int j0 = (int)(p.pos[1]*h_inv - offset);

    Vector2r base_coord(i0,j0);
    Vector2r fx = p.pos*h_inv - base_coord;

    real v0[2] {1.5-fx[0], 1.5-fx[1]};
    real v1[2] {fx[0]-1.,  fx[1]-1.};
    real v2[2] {fx[0]-.5,  fx[1]-.5};

    real w[3][2] = {{.5*v0[0]*v0[0],  .5*v0[1]*v0[1]},
                    {.75-v1[0]*v1[0], .75-v1[1]*v1[1]},
                    {.5*v2[0]*v2[0],  .5*v2[1]*v2[1]}};

    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
        {
            Vector2r dpos = Vector2r(i, j) - fx;
            real weight = w[i][0]*w[j][1];

            int idx_gridnode = i+i0 + (j+j0)*gridX;
            Vector2r node_velocity;
            node_velocity[0] = gprms.grid_array[1*nGridPitched + idx_gridnode];
            node_velocity[1] = gprms.grid_array[2*nGridPitched + idx_gridnode];
            p.velocity += weight * node_velocity;
            p.Bp += (4.*h_inv)*weight *(node_velocity*dpos.transpose());
        }

    // Advection
    p.pos += dt * p.velocity;

//    NACCUpdateDeformationGradient_q_hardening_2(p);
    NACCUpdateDeformationGradient_q_hardening(p);

    gprms.pts_array[icy::SimParams::posx*nPtsPitched + pt_idx] = p.pos[0];
    gprms.pts_array[icy::SimParams::posy*nPtsPitched + pt_idx] = p.pos[1];
    gprms.pts_array[icy::SimParams::velx*nPtsPitched + pt_idx] = p.velocity[0];
    gprms.pts_array[icy::SimParams::vely*nPtsPitched + pt_idx] = p.velocity[1];
    gprms.pts_array[icy::SimParams::Bp00*nPtsPitched + pt_idx] = p.Bp(0,0);
    gprms.pts_array[icy::SimParams::Bp01*nPtsPitched + pt_idx] = p.Bp(0,1);
    gprms.pts_array[icy::SimParams::Bp10*nPtsPitched + pt_idx] = p.Bp(1,0);
    gprms.pts_array[icy::SimParams::Bp11*nPtsPitched + pt_idx] = p.Bp(1,1);
    gprms.pts_array[icy::SimParams::Fe00*nPtsPitched + pt_idx] = p.Fe(0,0);
    gprms.pts_array[icy::SimParams::Fe01*nPtsPitched + pt_idx] = p.Fe(0,1);
    gprms.pts_array[icy::SimParams::Fe10*nPtsPitched + pt_idx] = p.Fe(1,0);
    gprms.pts_array[icy::SimParams::Fe11*nPtsPitched + pt_idx] = p.Fe(1,1);

    gprms.pts_array[icy::SimParams::idx_Jp*nPtsPitched + pt_idx] = p.Jp_inv;
    gprms.pts_array[icy::SimParams::idx_zeta*nPtsPitched + pt_idx] = p.zeta;

    // visualized variables
    gprms.pts_array[icy::SimParams::idx_p*nPtsPitched + pt_idx] = p.visualize_p;
    gprms.pts_array[icy::SimParams::idx_p0*nPtsPitched + pt_idx] = p.visualize_p0;
    gprms.pts_array[icy::SimParams::idx_q*nPtsPitched + pt_idx] = p.visualize_q;
    gprms.pts_array[icy::SimParams::idx_psi*nPtsPitched + pt_idx] = p.visualize_psi;
    gprms.pts_array[icy::SimParams::idx_case*nPtsPitched + pt_idx] = p.q;
}

//===========================================================================





__device__ Matrix2r dev(Matrix2r A)
{
    return A - A.trace()/2*Matrix2r::Identity();
}




// clamp x to range [a, b]
__device__ double clamp(double x, double a, double b)
{
    return max(a, min(b, x));
}


//===========================================================================

//===========================================================================

__device__ void svd(const real a[4], real u[4], real sigma[2], real v[4])
{
    GivensRotation<double> gv(0, 1);
    GivensRotation<double> gu(0, 1);
    singular_value_decomposition(a, gu, sigma, gv);
    gu.template fill<2, real>(u);
    gv.template fill<2, real>(v);
}

__device__ void svd2x2(const Matrix2r &mA, Matrix2r &mU, Matrix2r &mS, Matrix2r &mV)
{
    real U[4], V[4], S[2];
    real a[4] = {mA(0,0), mA(0,1), mA(1,0), mA(1,1)};
    svd(a, U, S, V);
    mU << U[0],U[1],U[2],U[3];
    mS << S[0],0,0,S[1];
    mV << V[0],V[1],V[2],V[3];
}


__device__ Matrix2r polar_decomp_R(const Matrix2r &val)
{
    // polar decomposition
    // http://www.cs.cornell.edu/courses/cs4620/2014fa/lectures/polarnotes.pdf
    real th = atan2(val(1,0) - val(0,1), val(0,0) + val(1,1));
    Matrix2r result;
    result << cosf(th), -sinf(th), sinf(th), cosf(th);
    return result;
}

__global__ void kernel_hello()
{
    printf("hello from CUDA\n");
}


void GPU_Implementation3::test()
{
    hipError_t err;
    kernel_hello<<<1,1,0,streamCompute>>>();
    err = hipGetLastError();

    if(err != hipSuccess)
    {
        std::cout << "cuda test error " << err << '\n';
        throw std::runtime_error("cuda test");
    }
    else
    {
        std::cout << "hello kernel executed successfully\n";
    }
    hipDeviceSynchronize();
}

void GPU_Implementation3::synchronize()
{
    if(!initialized) return;
    hipDeviceSynchronize();
}

