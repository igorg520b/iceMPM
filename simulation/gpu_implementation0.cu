#include "hip/hip_runtime.h"
#include "gpu_implementation0.h"
#include "parameters_sim.h"
#include "point.h"
#include "gridnode.h"
#include <stdio.h>
#include <iostream>

#include <Eigen/Core>
#include <Eigen/LU>



__constant__ icy::SimParams gprms;
__device__ icy::Point *gpu_points;
__device__ icy::GridNode *gpu_nodes;
__device__ int gpu_error_indicator;


GPU_Implementation0::GPU_Implementation0()
{
    test_cuda();
    hipEventCreate(&start);
    hipEventCreate(&stop);
}


void GPU_Implementation0::start_timing()
{
    hipEventRecord(start);
}

float GPU_Implementation0::end_timing()
{
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds;
}



void GPU_Implementation0::test_cuda()
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if(error_id) std::cout << "hipGetDeviceCount returs error " << error_id << '\n';
    std::cout << "CUDA devices " << deviceCount << '\n';

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    printf("Device \"%s\"\n", deviceProp.name);
    printf("Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

    Eigen::Matrix2f *results;
    hipMallocManaged(&results, sizeof(Eigen::Matrix2f)*3);
    Eigen::Matrix2f A;
    A << 1,7,-3,4;
    cuda_hello<<<1,1>>>(A, results);
    hipDeviceSynchronize();
    Eigen::Matrix2f U = results[0];
    Eigen::Matrix2f S = results[1];
    Eigen::Matrix2f V = results[2];

//    std::cout << "A=\n" << A << '\n';
//    std::cout << "U=\n" << U << '\n';
//    std::cout << "S=\n" << S << '\n';
//    std::cout << "V=\n" << V << '\n';
//    std::cout << "USV^T=\n" << U*S*V.transpose() << '\n';
    hipFree(results);

}





void GPU_Implementation0::cuda_update_constants(const icy::SimParams &prms)
{
    hipError_t err;
    int error_code = 0;
    err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_error_indicator), &error_code, sizeof(int));
    if(err != hipSuccess) throw std::runtime_error("gpu_error_indicator initialization");

    err = hipMemcpyToSymbol(HIP_SYMBOL(gprms), &prms, sizeof(icy::SimParams));
    if(err!=hipSuccess) throw std::runtime_error("cuda_update_constants: gprms");

    std::cout << "CUDA constants copied to device\n";
}

void GPU_Implementation0::cuda_allocate_arrays(size_t nGridNodes, size_t nPoints)
{
    hipFree((void*)gpu_points_);
    hipFree((void*)gpu_nodes_);

    hipError_t err;

    err = hipMalloc(&gpu_points_, sizeof(icy::Point)*nPoints);
    if(err != hipSuccess)
    {
        std::cout << "cuda_allocate_arrays can't allocate\n";
        throw std::runtime_error("cuda_allocate_arrays");
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_points), &gpu_points_, sizeof(gpu_points_));
    if(err != hipSuccess)
    {
        std::cout << "cuda_allocate_arrays hipMemcpyToSymbol error\n";
        throw std::runtime_error("cuda_allocate_arrays");
    }

    err = hipMalloc(&gpu_nodes_, sizeof(icy::GridNode)*nGridNodes);
    if(err != hipSuccess)
    {
        std::cout << "cuda_allocate_arrays can't allocate\n";
        throw std::runtime_error("cuda_allocate_arrays");
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_nodes), &gpu_nodes_, sizeof(gpu_nodes_));
    if(err != hipSuccess)
    {
        std::cout << "cuda_allocate_arrays hipMemcpyToSymbol error\n";
        throw std::runtime_error("cuda_allocate_arrays");
    }
    std::cout << "cuda_allocate_arrays done\n";

}

void GPU_Implementation0::transfer_ponts_to_device(size_t nPoints, void* hostSource)
{
    hipError_t err;
    err = hipMemcpy(gpu_points_, hostSource, nPoints*sizeof(icy::Point), hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        std::cout << "transfer_ponts_to_device failed with code \n";
        throw std::runtime_error("transfer_ponts_to_device");
    }
}

void GPU_Implementation0::cuda_transfer_from_device(size_t nPoints, void *hostArray)
{
    hipError_t err;
    err = hipMemcpy(hostArray, gpu_points_, nPoints*sizeof(icy::Point), hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        std::cout << "cuda_transfer_from_device failed\n";
        throw std::runtime_error("cuda_transfer_from_device");
    }

    int error_code = 0;
    err = hipMemcpyFromSymbol(&error_code, HIP_SYMBOL(gpu_error_indicator), sizeof(int));
    if(err != hipSuccess)
    {
        std::cout << "cuda_p2g hipMemcpyFromSymbol error\n";
        throw std::runtime_error("cuda_p2g");
    }
    if(error_code)
    {
        std::cout << "point is out of bounds\n";
        throw std::runtime_error("cuda_p2g");
    }
}

void GPU_Implementation0::cuda_device_synchronize()
{
    hipError_t err;
    err = hipDeviceSynchronize();
    if(err != hipSuccess)
    {
        std::cout << "cuda_device_synchronize failed\n";
        throw std::runtime_error("cuda_device_synchronize");
    }
}

void GPU_Implementation0::cuda_reset_grid(size_t nGridNodes)
{
    hipError_t err = hipMemsetAsync(gpu_nodes_, 0, sizeof(icy::GridNode)*nGridNodes);
    if(err != hipSuccess) throw std::runtime_error("cuda_reset_grid memset error");
}



void GPU_Implementation0::cuda_p2g(const int nPoints)
{
    hipError_t err;

    int blocksPerGrid = (nPoints + threadsPerBlock - 1) / threadsPerBlock;
    kernel_p2g<<<blocksPerGrid, threadsPerBlock>>>(nPoints);
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        std::cout << "cuda_p2g error executing kernel_p2g\n";
        throw std::runtime_error("cuda_p2g");
    }
}


void GPU_Implementation0::cuda_g2p(const int nPoints)
{
    hipError_t err;
    int blocksPerGrid = (nPoints + threadsPerBlock - 1) / threadsPerBlock;
    kernel_g2p<<<blocksPerGrid, threadsPerBlock>>>(nPoints);
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        std::cout << "cuda_g2p error\n";
        throw std::runtime_error("cuda_g2p");
    }
}


void GPU_Implementation0::cuda_update_nodes(const int nGridNodes,float indenter_x, float indenter_y)
{
    hipError_t err;
    int blocksPerGrid = (nGridNodes + threadsPerBlock - 1) / threadsPerBlock;
    kernel_update_nodes<<<blocksPerGrid, threadsPerBlock>>>(nGridNodes, indenter_x, indenter_y);
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        std::cout << "cuda_update_nodes\n";
        throw std::runtime_error("cuda_update_nodes");
    }
}






// ==============================  kernels  ====================================

__global__ void cuda_hello(Eigen::Matrix2f A, Eigen::Matrix2f *result)
{
    Eigen::Matrix2f &U = result[0];
    Eigen::Matrix2f &Sigma = result[1];
    Eigen::Matrix2f &V = result[2];
    svd2x2(A, U, Sigma, V);
    printf("Hello World from GPU!\n\n");
}

__device__ Eigen::Matrix2f polar_decomp_R(const Eigen::Matrix2f &val)
{
    // polar decomposition
    // http://www.cs.cornell.edu/courses/cs4620/2014fa/lectures/polarnotes.pdf
    float th = atan2f(val(1,0) - val(0,1), val(0,0) + val(1,1));
    Eigen::Matrix2f result;
    result << cosf(th), -sinf(th), sinf(th), cosf(th);
    return result;
}



__device__ float wqs(float x)
{
    x = fabsf(x);
    if (x < 0.5f) return -x * x + 3 / 4.0f;
    else if (x < 1.5f) return x * x / 2.0f - 3 * x / 2.0f + 9 / 8.0f;
    return 0;
}

__device__ float dwqs(float x)
{
    float x_abs = fabsf(x);
    if (x_abs < 0.5f) return -2.0f * x;
    else if (x_abs < 1.5f) return x - 3 / 2.0f * x / x_abs;
    return 0;
}

__device__ float wq(Eigen::Vector2f dx, double h)
{
    return wqs(dx[0]/h)*wqs(dx[1]/h);
}

__device__ Eigen::Vector2f gradwq(Eigen::Vector2f dx, double h)
{
    Eigen::Vector2f result;
    result[0] = dwqs(dx[0]/h)*wqs(dx[1]/h)/h;
    result[1] = wqs(dx[0]/h)*dwqs(dx[1]/h)/h;
    return result;
}


__global__ void kernel_p2g(const int nPoints)
{
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(pt_idx >= nPoints) return;

    const float &particle_volume = gprms.ParticleVolume;
    const float &cellsize = gprms.cellsize;
    const float &Dp_inv = gprms.Dp_inv;
    const float &lambda = gprms.lambda;
    const float &mu = gprms.mu;
    const int &gridX = gprms.GridX;
    const int &gridY = gprms.GridY;
    const float &particle_mass = gprms.ParticleMass;

    icy::Point &p = gpu_points[pt_idx];

    // NACC constitutive model
    Eigen::Matrix2f Re = polar_decomp_R(p.Fe);
    float Je = p.Fe.determinant();
    Eigen::Matrix2f dFe = 2.f * mu*(p.Fe - Re)* p.Fe.transpose() +
            lambda * (Je - 1.f) * Je * Eigen::Matrix2f::Identity();
    Eigen::Matrix2f Ap = dFe * particle_volume;

    // continue with distributing to the grid
    constexpr float offset = 0.5f;  // 0 for cubic; 0.5 for quadratic
    const int i0 = (int)(p.pos[0]/cellsize - offset);
    const int j0 = (int)(p.pos[1]/cellsize - offset);

    for (int i = i0; i < i0+3; i++)
        for (int j = 0; j < j0+3; j++)
        {
            int idx_gridnode = i + j*gridX;
            if(i < 0 || j < 0 || i >=gridX || j>=gridY || idx_gridnode < 0)
                gpu_error_indicator = 1;

            Eigen::Vector2f pos_node(i*cellsize, j*cellsize);
            Eigen::Vector2f d = p.pos - pos_node;
            float Wip = wq(d, cellsize);   // weight
            Eigen::Vector2f dWip = gradwq(d, cellsize);    // weight gradient

            // APIC increments
            float incM = Wip * particle_mass;
            Eigen::Vector2f incV = incM * (p.velocity + Dp_inv * p.Bp * (-d));
            Eigen::Vector2f incFi = Ap * dWip;

            // Udpate mass, velocity and force
            icy::GridNode &gn = gpu_nodes[idx_gridnode];
            atomicAdd(&gn.mass, incM);
            atomicAdd(&gn.velocity[0], incV[0]);
            atomicAdd(&gn.velocity[1], incV[1]);
            atomicAdd(&gn.force[0], incFi[0]);
            atomicAdd(&gn.force[1], incFi[1]);
        }
}

__global__ void kernel_update_nodes(const int nGridNodes, float indenter_x, float indenter_y)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= nGridNodes) return;

    icy::GridNode &gn = gpu_nodes[idx];
    if(gn.mass == 0) return;

    const float &gravity = gprms.Gravity;
    const float &indRsq = gprms.IndRSq;
    const int &gridX = gprms.GridX;
    const int &gridY = gprms.GridY;
    const float &dt = gprms.InitialTimeStep;
    const float &ind_velocity = gprms.IndVelocity;
    const float &cellsize = gprms.cellsize;
    const float &ice_friction_coeff = gprms.IceFrictionCoefficient;

    const Eigen::Vector2f gravity_(0,-gravity);
    const Eigen::Vector2f vco(ind_velocity,0);  // velocity of the collision object (indenter)
    const Eigen::Vector2f indCenter(indenter_x, indenter_y);

    gn.velocity = gn.velocity/gn.mass + dt*(-gn.force/gn.mass + gravity_);

    int idx_x = idx % gridX;
    int idx_y = idx / gridX;

    // indenter
    Eigen::Vector2f gnpos(idx_x*cellsize, idx_y*cellsize);
    Eigen::Vector2f n = gnpos - indCenter;
    if(n.squaredNorm() < indRsq)
    {
        // grid node is inside the indenter
        Eigen::Vector2f vrel = gn.velocity - vco;
        n.normalize();
        float vn = vrel.dot(n);   // normal component of the velocity
        if(vn < 0)
        {
            Eigen::Vector2f vt = vrel - n*vn;   // tangential portion of relative velocity
            gn.velocity = vco + vt + ice_friction_coeff*vn*vt.normalized();
        }
    }

    // attached bottom layer
    if(idx_y <= 3) gn.velocity.setZero();
    else if(idx_y >= gridY-4 && gn.velocity[1]>0) gn.velocity[1] = 0;
    if(idx_x <= 3 && gn.velocity.x()<0) gn.velocity[0] = 0;
    else if(idx_x >= gridX-5) gn.velocity[0] = 0;
}

__global__ void kernel_g2p(const int nPoints)
{
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(pt_idx >= nPoints) return;

    icy::Point &p = gpu_points[pt_idx];

    const float &cellsize = gprms.cellsize;
    const float &dt = gprms.InitialTimeStep;
    const int &gridX = gprms.GridX;

    p.velocity.setZero();
    p.Bp.setZero();

    constexpr float offset = 0.5f;  // 0 for cubic; 0.5 for quadratic
    const int i0 = (int)((p.pos[0])/cellsize - offset);
    const int j0 = (int)((p.pos[1])/cellsize - offset);
    const Eigen::Vector2f pointPos_copy = p.pos;
    p.pos.setZero();

    Eigen::Matrix2f T;
    T.setZero();

    for (int i = i0; i < i0+3; i++)
        for (int j = j0; j < j0+3; j++)
        {
            const int idx_gridnode = i + j*gridX;
            const icy::GridNode &node = gpu_nodes[idx_gridnode];

            Eigen::Vector2f pos_node(i*cellsize, j*cellsize);
            Eigen::Vector2f d = pointPos_copy - pos_node;   // dist
            float Wip = wq(d, cellsize);   // weight
            Eigen::Vector2f dWip = gradwq(d, cellsize);    // weight gradient

            p.velocity += Wip * node.velocity;
            p.Bp += Wip *(node.velocity*(-d).transpose());
            // Update position and nodal deformation
            p.pos += Wip * (pos_node + dt * node.velocity);
            T += node.velocity * dWip.transpose();
        }
    NACCUpdateDeformationGradient(p, T);
}


__device__ void NACCUpdateDeformationGradient(icy::Point &p, Eigen::Matrix2f &FModifier)
{
    constexpr float magic_epsilon = 1e-5f;
    constexpr int d = 2; // dimensions
    float &alpha = p.NACC_alpha_p;
    const float &mu = gprms.mu;
    const float &kappa = gprms.kappa;
    const float &beta = gprms.NACC_beta;
    const float &M_sq = gprms.NACC_M_sq;
    const float &xi = gprms.NACC_xi;
    const bool &hardening = gprms.NACC_hardening;
    const float &dt = gprms.InitialTimeStep;

    Eigen::Matrix2f FeTr = (Eigen::Matrix2f::Identity() + dt * FModifier) * p.Fe;

    Eigen::Matrix2f U, V, Sigma;

    svd2x2(FeTr, U, Sigma, V);

    // line 4
    float p0 = kappa * (magic_epsilon + sinhf(xi * fmaxf(-alpha, 0.f)));

    // line 5
    float Je_tr = Sigma(0,0)*Sigma(1,1);    // this is for 2D

    // line 6
    Eigen::Matrix2f SigmaSquared = Sigma*Sigma;
    Eigen::Matrix2f SigmaSquaredDev = SigmaSquared - SigmaSquared.trace()/2.f*Eigen::Matrix2f::Identity();
    float J_power_neg_2_d_mulmu = mu * powf(Je_tr, -2.f / (float)d);///< J^(-2/dim) * mu
    Eigen::Matrix2f s_hat_tr = J_power_neg_2_d_mulmu * SigmaSquaredDev;

    // line 7
    float psi_kappa_partial_J = (kappa/2.f) * (Je_tr - 1.f / Je_tr);

    // line 8
    float p_trial = -psi_kappa_partial_J * Je_tr;

    // line 9 (case 1)
    float y = (1.f + 2.f*beta)*(3.f-(float)d/2.f)*s_hat_tr.norm() + M_sq*(p_trial + beta*p0)*(p_trial - p0);
    if(p_trial > p0)
    {
        float Je_new = sqrtf(-2.f*p0 / kappa + 1.f);
        Eigen::Matrix2f Sigma_new = Eigen::Matrix2f::Identity() * powf(Je_new, 1.f/(float)d);
        p.Fe = U*Sigma_new*V.transpose();
        if(hardening) alpha += logf(Je_tr / Je_new);
    }

    // line 14 (case 2)
    else if(p_trial < -beta*p0)
    {
        float Je_new = sqrtf(2.f*beta*p0/kappa + 1.f);
        Eigen::Matrix2f Sigma_new = Eigen::Matrix2f::Identity() * pow(Je_new, 1.f/(float)d);
        p.Fe = U*Sigma_new*V.transpose();
        if(hardening) alpha += logf(Je_tr / Je_new);
    }

    // line 19 (case 3)
    else if(y >= magic_epsilon*10)
    {
        if(hardening && p0 > magic_epsilon && p_trial < p0 - magic_epsilon && p_trial > -beta*p0 + magic_epsilon)
        {
            float p_c = (1.f-beta)*p0/2.f;  // line 23
            float q_tr = sqrtf(3.f-d/2.f)*s_hat_tr.norm();   // line 24
            Eigen::Vector2f direction(p_c-p_trial, -q_tr);  // line 25
            direction.normalize();
            float C = M_sq*(p_c-beta*p0)*(p_c-p0);
            float B = M_sq*direction[0]*(2.f*p_c-p0+beta*p0);
            float A = M_sq*direction[0]*direction[0]+(1.f+2.f*beta)*direction[1]*direction[1];  // line 30
            float l1 = (-B+sqrtf(B*B-4.f*A*C))/(2.f*A);
            float l2 = (-B-sqrtf(B*B-4.f*A*C))/(2.f*A);
            float p1 = p_c + l1*direction[0];
            float p2 = p_c + l2*direction[0];
            float p_x = (p_trial-p_c)*(p1-p_c) > 0 ? p1 : p2;
            float Je_x = sqrtf(fabsf(-2.f*p_x/kappa + 1.f));
            if(Je_x > magic_epsilon*10) alpha += logf(Je_tr / Je_x);
        }

        float expr_under_root = (-M_sq*(p_trial+beta*p0)*(p_trial-p0))/((1+2.f*beta)*(3.f-d/2.));
        Eigen::Matrix2f B_hat_E_new = sqrtf(expr_under_root)*(powf(Je_tr,2.f/d)/mu)*s_hat_tr.normalized() +
                Eigen::Matrix2f::Identity()*SigmaSquared.trace()/(float)d;
        Eigen::Matrix2f Sigma_new;
        Sigma_new << sqrt(B_hat_E_new(0,0)), 0,
                0, sqrt(B_hat_E_new(1,1));
        p.Fe = U*Sigma_new*V.transpose();
    }
    else
    {
        p.Fe = FeTr;
    }
    //p.visualized_value = alpha;
}



/**
\brief 2x2 SVD (singular value decomposition) a=USV'
\param[in] a Input matrix.
\param[out] u Robustly a rotation matrix.
\param[out] sigma Vector of singular values sorted with decreasing magnitude. The second one can be negative.
\param[out] v Robustly a rotation matrix.
*/
__device__ void svd(const float a[4], float u[4], float sigma[2], float v[4])
{
    GivensRotation<double> gv(0, 1);
    GivensRotation<double> gu(0, 1);
    singular_value_decomposition(a, gu, sigma, gv);
    gu.template fill<2, float>(u);
    gv.template fill<2, float>(v);
}


__device__ void svd2x2(const Eigen::Matrix2f &mA,
                       Eigen::Matrix2f &mU,
                       Eigen::Matrix2f &mS,
                       Eigen::Matrix2f &mV)
{
    float U[4], V[4], S[2];
    float a[4] = {mA(0,0), mA(0,1), mA(1,0), mA(1,1)};
    svd(a, U, S, V);

    mU << U[0],U[1],U[2],U[3];
    mS << S[0],0,0,S[1];
    mV << V[0],V[1],V[2],V[3];
}

